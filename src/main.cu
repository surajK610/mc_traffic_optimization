// main.cpp
#include <cmath>
#include <iostream>

#include "anneal/anneal_csa.hpp"
#include "anneal/anneal_csa_st.hpp"
#include "anneal/anneal_msa.hpp"
#include "anneal/anneal_msa_st.hpp"
#include "anneal/context.hpp"
#include "schwefel/schwefel.hpp"
#include "schwefel/schwefel_cuda.hpp"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include <sys/time.h>

void progress(
    void* instance, double cost, float tgen, float tacc, int opt_id, int iter)
{
    printf(
        "bestcost=%1.3e \t tgen=%1.3e \t tacc=%1.3e \t thread=%d\n",
        cost,
        tgen,
        tacc,
        opt_id);
    return ;
}




int main(int argc, char** argv)
{
    struct timeval start;
    struct timeval end;

    int optionSA = 0;
    int optionCU = 0;
    if (argc > 1)
        optionSA = atoi(argv[1]);
    if (argc > 2)
        optionCU = atoi(argv[2]);

    BaseSolver* solver = nullptr;
    if (optionSA == OPTION_MSA_ST) {
        solver = new MSA_ST::SolverMultipleST();
    } else if (optionSA == OPTION_MSA) {
        solver = new MSA::SolverMultiple();
    } else if (optionSA == OPTION_CSA_ST) {
        solver = new CSA_ST::SolverCoupledST();
    } else if (optionSA == OPTION_CSA) {
        solver = new CSA::SolverCoupled();
    } else {
        std::cout << "Invalid option for simulated annealing" << std::endl;
        return EXIT_FAILURE;
    }

    gettimeofday(&start, 0);
    if (optionCU == 0) {
        srand(0);
        double* x = new double[SCHWEFEL::DIM];
        for (int i = 0; i < SCHWEFEL::DIM; ++i)
            x[i] = drand48();

        double cost = SCHWEFEL::f(nullptr, x);
        printf("Initial cost: %f\n", cost);
        
        solver->minimize(SCHWEFEL::DIM, x, f, step, progress, nullptr);

        cost = f(nullptr, x);
        printf("Best cost: %f\nx =\n", cost);

        for (int i = 0; i < SCHWEFEL::DIM; ++i)
            std::cout << 500 * x[i] << " ";
            
        std::cout << std::endl;
        delete[] x;
        gettimeofday(&end, 0);
        std::cout << "Took " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " microseconds" << std::endl;
        
        return EXIT_SUCCESS;
    } else if (optionCU == 1) {
        srand(0);
        double* x = new double[SCHWEFEL::DIM];
        for (int i = 0; i < SCHWEFEL::DIM; ++i)
            x[i] = drand48();

        dim3 blockSize(256, 1, 1);
        dim3 numBlocks((SCHWEFEL::DIM + blockSize.x - 1) / blockSize.x, 1, 1);

        // setup_kernel<<<numBlocks, blockSize>>>(devStates, time(NULL));

        double cost = f_c(nullptr, x);
        printf("Initial cost: %f\n", cost);

        solver->minimize(SCHWEFEL::DIM, x,  f_c, step_c, progress, nullptr);
        cost = f_c(nullptr, x);

        printf("Best cost: %f\nx =\n", cost);

        for (int i = 0; i <  SCHWEFEL::DIM; ++i)
            std::cout << 500 * x[i] << " ";
            
        std::cout << std::endl;

        delete[] x;
        gettimeofday(&end, 0);
        std::cout << "Took " << (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_usec - start.tv_usec) << " microseconds" << std::endl;
        return EXIT_SUCCESS;
    } else {
        std::cout << "Invalid option for CUDA" << std::endl;
        return EXIT_FAILURE;
    }
    
}
