#include "hip/hip_runtime.h"
// schwefel_cuda.hpp file

#ifndef SCHWEFEL_CUDA
#define SCHWEFEL_CUDA

#include "schwefel_cuda.hpp"
#include <cmath>
#include <stdio.h>
#include <iostream>
#include <typeinfo>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

const int DIM = 10;

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx < DIM)
      hiprand_init(seed, idx, 0, &state[idx]); 
}

__global__ void f_kernel(double* x, double* result) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < DIM) {
      result[idx] = 500 * x[idx] * sin(sqrt(fabs(500 * x[idx])));
    }
}


__global__ void f_kernel_warp(double* x, double* result) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t warp_id = threadIdx.x / 32;
    size_t lane = threadIdx.x % 32;
    if (idx < DIM) {
      result[idx] = 500 * x[idx] * sin(sqrt(fabs(500 * x[idx])));

      syncwarp();
      for (int i = 16; i >= 1; i /= 2) {
        double temp = __shfl_down_sync(0xFFFFFFFF, result[idx], i);
        if (lane < i) {
          result[idx] += temp;
        }
      }

    }
}

__global__ void step_kernel(double* y, const double* x, float tgen, hiprandState *state) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  if (idx < DIM) {
    // hiprand_init(0, idx, 0, &state[idx]); 
    hiprandState localState = state[idx];         
    double randVal = hiprand_uniform(&localState); 
    y[idx] = fmod(x[idx] + tgen * tanf(M_PI * (randVal - 0.5)), 1.0);
    state[idx] = localState;                      
  }
}

double f_c(void* instance, double* x) {
  double *d_x, *d_result;
  double* h_result = new double[DIM];

  hipMalloc((void**) &d_x, DIM * sizeof(double));
  hipMalloc((void**) &d_result, DIM * sizeof(double));
  hipMemcpy(d_x, x, DIM * sizeof(double), hipMemcpyHostToDevice);

  dim3 blockSize(256, 1, 1);
  dim3 numBlocks((DIM + blockSize.x - 1) / blockSize.x, 1, 1);
  f_kernel<<<numBlocks, blockSize>>>(d_x, d_result);
  cudaSynchronize();
  // fflush(stdout);
  hipMemcpy(h_result, d_result, DIM * sizeof(double), hipMemcpyDeviceToHost);

  double sum = 0.;
  // printf("f_c: ");
  for (int i = 0; i < DIM; ++i) {
    // printf("%f ", h_result[i]);
    sum += h_result[i];
  }
  // printf("\n");
  delete[] h_result;
  hipFree(d_x);
  hipFree(d_result);

  return 418.9829 * DIM - sum;
}

void step_c(void* instance, double* y, const double* x, float tgen) {
  static bool is_setup_done = false;
  static hiprandState *dev_states;

  double *d_x, *d_y;

  hipMalloc((void**) &d_x, DIM * sizeof(double));
  hipMalloc((void**) &d_y, DIM * sizeof(double));
  hipMemcpy(d_x, x, DIM * sizeof(double), hipMemcpyHostToDevice);

  dim3 blockSize(256, 1, 1);
  dim3 numBlocks((DIM + blockSize.x - 1) / blockSize.x, 1, 1);

  if (!is_setup_done) {
    hipMalloc((void**) &dev_states, DIM * sizeof(hiprandState));
    setup_kernel<<<numBlocks, blockSize>>>(dev_states, time(NULL));
    is_setup_done = true;
    printf("Setup done\n");
  }
  step_kernel<<<numBlocks, blockSize>>>(d_y, d_x, tgen, dev_states);
  cudaSynchronize();
  hipMemcpy(y, d_y, DIM * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);

}


#endif