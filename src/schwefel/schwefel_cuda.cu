#include "hip/hip_runtime.h"
// schwefel_cuda.hpp file

#ifndef SCHWEFEL_CUDA
#define SCHWEFEL_CUDA

#include "schwefel_cuda.hpp"
#include <cmath>
#include <stdio.h>
#include <iostream>
#include <typeinfo>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

const int DIM = 10;
const int WARP_SIZE = 32;
const int FULL_MASK = 0xffffffff;
const bool WARP = false;

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx < DIM)
      hiprand_init(seed, idx, 0, &state[idx]); 
}

__global__ void f_kernel(double* x, double* result) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < DIM) {
      result[idx] = 500 * x[idx] * sin(sqrt(fabs(500 * x[idx])));
    }
}

__global__ void f_kernel_warp(double* x, double* result) {
  // max 1 block
    size_t lane = threadIdx.x % WARP_SIZE;
    int warpid = threadIdx.x/WARP_SIZE;
    int nwarps = blockDim.x/WARP_SIZE;

    double sum = 0.0;
    if (lane < DIM) {
      for (size_t idx = lane + WARP_SIZE*warpid; idx < DIM; idx += WARP_SIZE*nwarps) { // modulus addition
        if (idx < DIM) {
          sum += 500 * x[idx] * sin(sqrt(fabs(500 * x[idx])));
        }
      }
      __syncwarp();
      for (size_t offset = WARP_SIZE/2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(FULL_MASK, sum, offset);
      }

      __shared__ double s_mem[1024/WARP_SIZE];
      if (lane == 0) {
        s_mem[warpid] = sum;
      }

      __syncthreads(); // sync threads within block
      if (threadIdx.x == 0) { // first lane in first warp
        for (int j = 0; j < nwarps; ++j) {
          result[0] += s_mem[j];
        }   
      }
    }
}

__global__ void step_kernel(double* y, const double* x, float tgen, hiprandState *state) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x; 
  if (idx < DIM) {
    // hiprand_init(0, idx, 0, &state[idx]); 
    hiprandState localState = state[idx];         
    double randVal = hiprand_uniform(&localState); 
    y[idx] = fmod(x[idx] + tgen * tanf(M_PI * (randVal - 0.5)), 1.0);
    state[idx] = localState;                      
  }
}

double f_c(void* instance, double* x) {
  double *d_x, *d_result;
  double* h_result = new double[DIM];

  hipMalloc((void**) &d_x, DIM * sizeof(double));
  hipMalloc((void**) &d_result, DIM * sizeof(double));
  hipMemcpy(d_x, x, DIM * sizeof(double), hipMemcpyHostToDevice);

  dim3 blockSize(256, 1, 1);
  dim3 numBlocks((DIM + blockSize.x - 1) / blockSize.x, 1, 1);
  double sum = 0.0;

  if (WARP) {
    if (DIM > 1024) {
      printf("Warp kernel only supports DIM <= 1024\n");
      exit(1);
    }
    
    f_kernel_warp<<<1, 1024>>>(d_x, d_result); // only 1 block, max 1024 threads
    hipMemcpy(h_result, d_result, 1 * sizeof(double), hipMemcpyDeviceToHost);
    sum = h_result[0];
  } else {
    f_kernel<<<numBlocks, blockSize>>>(d_x, d_result);
    hipMemcpy(h_result, d_result, DIM * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < DIM; ++i) {
      sum += h_result[i];
    }
  }
  hipDeviceSynchronize();
  // fflush(stdout);

  
  delete[] h_result;
  hipFree(d_x);
  hipFree(d_result);

  return 418.9829 * DIM - sum;
}

void step_c(void* instance, double* y, const double* x, float tgen) {
  static bool is_setup_done = false;
  static hiprandState *dev_states;

  double *d_x, *d_y;

  hipMalloc((void**) &d_x, DIM * sizeof(double));
  hipMalloc((void**) &d_y, DIM * sizeof(double));
  hipMemcpy(d_x, x, DIM * sizeof(double), hipMemcpyHostToDevice);

  dim3 blockSize(256, 1, 1);
  dim3 numBlocks((DIM + blockSize.x - 1) / blockSize.x, 1, 1);

  if (!is_setup_done) {
    hipMalloc((void**) &dev_states, DIM * sizeof(hiprandState));
    setup_kernel<<<numBlocks, blockSize>>>(dev_states, time(NULL));
    is_setup_done = true;
    printf("Setup done\n");
  }
  step_kernel<<<numBlocks, blockSize>>>(d_y, d_x, tgen, dev_states);
  hipDeviceSynchronize();
  hipMemcpy(y, d_y, DIM * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);

}


#endif