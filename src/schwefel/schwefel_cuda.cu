// schwefel_cuda.cu file

#ifndef SCHWEFEL_CUDA_HPP
#define SCHWEFEL_CUDA_HPP

#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <iostream>
#include <typeinfo>
#include <limits>

/**
 * Variable Explanations for CUDA Simulated Annealing Algorithm
 *
 * 1. x: Input array for the Schwefel function or the annealing step.
 *    - Type: double*
 *    - Size: 'dim' elements, where each element is a double.
 *    - Role: In the context of the Schwefel function, it represents the point in the function's domain at which the function is evaluated. In the context of the annealing step, it represents the current state of the optimization variable.
 *
 * 2. y: Output array for the annealing step.
 *    - Type: double*
 *    - Size: 'dim' elements, similar to 'x'.
 *    - Role: Stores the new state of the optimization variable after performing the annealing step. Each element of 'y' is computed based on the corresponding element of 'x', with some perturbation determined by 'tgen' and a random number.
 *
 * 3. tgen (generation temperature): Control parameter for the annealing step.
 *    - Type: float
 *    - Role: Determines the variance of the distribution from which the perturbation is drawn during the annealing step. A higher 'tgen' allows for larger jumps in the solution space, facilitating exploration.
 *
 * 4. dim (dimension): The size of the optimization problem.
 *    - Type: int
 *    - Role: Specifies the number of variables in the optimization problem, which directly determines the size of 'x' and 'y' arrays.
 *
 * 5. curandState: Array of states for the CURAND random number generator.
 *    - Type: curandState*
 *    - Size: One state per thread. The total number is usually equal to the total number of threads launched in the kernel.
 *    - Role: These states are used to generate random numbers in a parallel and efficient manner on the GPU. Each thread uses its own state to ensure the independence of random numbers across threads.
 *
 * 6. dev_x, dev_y, dev_result: Device-side counterparts of 'x', 'y', and the intermediate result array.
 *    - Type: double*
 *    - Size: Same as their host-side counterparts ('x', 'y', and an array for storing intermediate results of the Schwefel function).
 *    - Role: These are the device-side (GPU memory) arrays. 'dev_x' and 'dev_y' are used in the annealing step, while 'dev_result' is used to store the output of the Schwefel function computation on the GPU before transferring it back to the host.
 *
 * Note: The size of 'curandState' and the number of threads per block should be carefully chosen to balance efficient use of GPU resources and performance. The block size is often set to a multiple of 32 to align with the warp size of the GPU, enhancing efficiency.
 */

// Setup kernel for initializing CURAND states
__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate the global thread index
    hiprand_init(seed, idx, 0, &state[idx]);           // Initialize CURAND state for each thread
}

// Kernel to compute the Schwefel function for each element
__global__ void f_kernel(double* x, double* result, int dim) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate the global thread index
    if (idx < dim) {
        // Compute the Schwefel function for the element at idx
        result[idx] = 500 * x[idx] * sin(sqrt(fabs(500 * x[idx])));
    }
}

// Kernel for the annealing step
__global__ void step_kernel(double* y, const double* x, float tgen, int dim, hiprandState *state) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate the global thread index
    if (idx < dim) {
        hiprandState localState = state[idx];          // Get the local CURAND state
        double randVal = hiprand_uniform(&localState); // Generate a random number
        // Perform the annealing step
        y[idx] = fmod(x[idx] + tgen * tan(M_PI * (randVal - 0.5)), 1.0);
        state[idx] = localState;                      // Update the CURAND state
    }
}

// Host function to compute the Schwefel function
double f_c(void* instance, double* x, int dim, double* dev_x, double* dev_result) {
    // Copy data from host to device
    hipMemcpy(dev_x, x, dim * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;  // Block size, chosen for efficient GPU utilization
    int numBlocks = (dim + blockSize - 1) / blockSize;  // Calculate the number of blocks needed
    // Launch the kernel to compute the Schwefel function
    f_kernel<<<numBlocks, blockSize>>>(dev_x, dev_result, dim);

    double* host_result = new double[dim];
    // Copy the result back from device to host
    hipMemcpy(host_result, dev_result, dim * sizeof(double), hipMemcpyDeviceToHost);

    double sum = 0.;
    // Accumulate the results to compute the final value of the function
    for (int i = 0; i < dim; ++i) {
        sum += host_result[i];
    }

    delete[] host_result;  // Free the host memory

    return 418.9829 * dim - sum;  // Return the computed value
}

// Host function for the annealing step
void step_c(void* instance, double* y, const double* x, float tgen, int dim, double* dev_x, double* dev_y, hiprandState *dev_states) {
    // Copy data from host to device
    hipMemcpy(dev_x, x, dim * sizeof(double), hipMemcpyHostToDevice);

    int blockSize = 256;  // Block size, chosen for efficient GPU utilization
    int numBlocks = (dim + blockSize - 1) / blockSize;  // Calculate the number of blocks needed
    // Launch the kernel for the annealing step
    step_kernel<<<numBlocks, blockSize>>>(dev_y, dev_x, tgen, dim, dev_states);

    // Copy the result back from device to host
    hipMemcpy(y, dev_y, dim * sizeof(double), hipMemcpyDeviceToHost);
}


/////////////////////////////////////////////////////////////////
/// Merged Kernel for Annealing Step and Cost Computation
/////////////////////////////////////////////////////////////////

// Merged Kernel for Annealing Step and Cost Computation
__global__ void step_cost_kernel(double* y, const double* x, float tgen, int dim, hiprandState* state, double* result) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < dim) {
        // Generate a random number for the annealing step
        hiprandState localState = state[idx];
        double randVal = hiprand_uniform(&localState);
        // Perform the annealing step
        double newY = fmod(x[idx] + tgen * tan(M_PI * (randVal - 0.5)), 1.0);
        y[idx] = newY;  // Update the new state
        // Compute the Schwefel function for the new state
        result[idx] = 500 * newY * sin(sqrt(fabs(500 * newY)));
    }
}

// Final computation and reduction of the Schwefel function
__global__ void final_cost_reduction(double* result, int dim) {
    double sum = 0.0;
    for (int i = 0; i < dim; ++i) {
        sum += result[i];
    }
    result[0] = 418.9829 * dim - sum; // Store the final cost in the first element
}

// Host function to perform the annealing step and compute the cost
void step_cost_c(void* instance, double* y, const double* x, float tgen, int dim, double* dev_x, double* dev_y, hiprandState* dev_states, double* dev_result, hipStream_t stream) {
    // Copy data from host to device
    hipMemcpyAsync(dev_x, x, dim * sizeof(double), hipMemcpyHostToDevice, stream);

    // Calculate the number of blocks and threads
    int blockSize = 256;  // Chosen for efficient GPU utilization
    int numBlocks = (dim + blockSize - 1) / blockSize;

    // Launch the merged kernel for the annealing step and partial cost computation
    step_cost_kernel<<<numBlocks, blockSize, 0, stream>>>(dev_y, dev_x, tgen, dim, dev_states, dev_result);

    // Final reduction to compute the total cost
    final_cost_reduction<<<1, 1, 0, stream>>>(dev_result, dim);

    // Copy the result back from device to host asynchronously
    hipMemcpyAsync(y, dev_y, dim * sizeof(double), hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync((void*)&x[0], dev_result, sizeof(double), hipMemcpyDeviceToHost, stream); // Copy only the final cost
}

#endif

int main(int argc, char *argv[]) {
    
    // Problem configuration
    int dim = 1024;  // Dimension of the optimization problem
    float tgen = 0.1f;  // Temperature for simulated annealing
    const int iterations = 100;  // Number of iterations for testing

    std::cout << "Data types:\n";
    std::cout << "dim type: " << typeid(dim).name() << "\n";
    std::cout << "tgen type: " << typeid(tgen).name() << "\n";

    // Host memory allocation
    double *h_x = new double[dim];
    double *h_y = new double[dim];
    double f_result;

    // Initialize input array with random values
    std::cout << "Initializing input array...\n";
    // for (int i = 0; i < dim; ++i) {
    //     h_x[i] = static_cast<double>(rand()) / RAND_MAX;
    //     std::cout << "h_x[" << i << "] = " << h_x[i] << "\n";
    // }

    // Device memory allocation
    double *d_x, *d_y, *d_result;
    hipMalloc(&d_x, dim * sizeof(double));
    hipMalloc(&d_y, dim * sizeof(double));
    hipMalloc(&d_result, dim * sizeof(double));

    // CURAND state initialization
    hiprandState *d_states;
    hipMalloc(&d_states, dim * sizeof(hiprandState));

    // Setup CURAND states
    int blockSize = 256;
    int numBlocks = (dim + blockSize - 1) / blockSize;
    setup_kernel<<<numBlocks, blockSize>>>(d_states, time(NULL));
    hipDeviceSynchronize();  // Wait for kernel completion

    for (int iter = 0; iter < iterations; ++iter) {
        std::cout << "Iteration " << iter + 1 << "...\n";

        // Perform annealing step
        std::cout << "Performing annealing step...\n";
        step_c(nullptr, h_y, h_x, tgen, dim, d_x, d_y, d_states);
        std::cout << "Annealing step completed.\n";

        // Compute Schwefel function
        std::cout << "Computing Schwefel function...\n";
        f_result = f_c(nullptr, h_x, dim, d_x, d_result);
        std::cout << "Schwefel function computed.\n";

        // Print the result
        std::cout << "Result of f_c after iteration " << iter + 1 << ": " << f_result << "\n";
    }

    // Verification (Optional): Check and print some values from h_y
    std::cout << "Verifying output array (h_y):\n";
    for (int i = 0; i < std::min(dim, 10); ++i) {  // Print first 10 values as a sample
        std::cout << "h_y[" << i << "] = " << h_y[i] << "\n";
    }

    // Free resources
    delete[] h_x;
    delete[] h_y;
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    hipFree(d_states);

    return 0;
}